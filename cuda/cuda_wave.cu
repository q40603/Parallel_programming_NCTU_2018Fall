/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                     /* number of time steps */
    tpoints,                /* total points along string */
    rcode;                      /* generic return code */
float  values[MAXPOINTS];     /* values at time t */

/**********************************************************************
 *  Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
__global__
void init(float *values, int num_of_points){
   int index = blockIdx.x*blockDim.x + threadIdx.x;
   float x;
   //fac = 2.0 * PI;
   //tmp = num_of_points;
   x = 1.0*(float)index / (float)num_of_points;
   values[index] = sin (6.2831853 * x);
}
void init_line(void)
{
   int j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0;
   tmp = tpoints - 1;
   for (j = 0; j < tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   }
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__global__
void do_math(float *values, int num_of_points, int num_of_iterations)
{
   int index = blockIdx.x*blockDim.x + threadIdx.x;
   if (index < num_of_points && index > 0)
   {
      float old_val, val, new_val;
      old_val = val = values[index];
      for (int i = 0; i < num_of_iterations; i++)
      {
          new_val = (1.82 * val) - old_val;
          old_val = val;
          val = new_val;
      }
      values[index] = val;
   }
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update()
{

   float *arr;
   int size = tpoints * sizeof(float);
   hipMalloc((void**)&arr, size);
   hipMemcpy(arr, values, size, hipMemcpyHostToDevice);
   /* Update values for each time step */
   do_math<<<((tpoints + 1023) >> 10), 1024>>>(arr, tpoints, nsteps);
   hipMemcpy(values, arr, size, hipMemcpyDeviceToHost);
   hipFree(arr);
   values[0] = values[tpoints-1] = 0.0;
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 0; i < tpoints; i++) {
      printf("%6.4f ", values[i]);
      if ((i+1)%10 == 0)
         printf("\n");
   }
}
void output_data(int n,int point){
   char filename[100];
   FILE *fp;
   sprintf(filename, "output_%d.txt", n);
   fp = fopen(filename, "w");
   fprintf(fp, "%d\n", point);
   for (int i = 0; i < tpoints; i++) {
      fprintf(fp, "%6.4f ", values[i]);
   }
   fprintf(fp, "\n");

   fclose(fp); 
}
/**********************************************************************
 * Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
   sscanf(argv[1],"%d",&tpoints);
   sscanf(argv[2],"%d",&nsteps);
   check_param();
   printf("Initializing points on the line...\n");
   init_line();
   printf("Updating all points for all time steps...\n");
   update();
   printf("Printing final results...\n");
   printfinal();
   //output_data(1,tpoints);
   printf("\nDone.\n\n");
   
   return 0;
}